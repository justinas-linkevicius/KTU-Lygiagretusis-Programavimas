﻿/* Justinas Linkevicius
* IFF-3/2
* L1c
*

1. Kokia tvarka startuoja procesai?
****  tokia, kokia užrašyti

2. Kokia tvarka vykdomi procesai?
**** tokia, kokia startuoja

3. Kiek iteracijų iš eilės padaro vienas procesas?
**** vieną pilna

4. Kokia tvarka atspausdinami to paties masyvo duomenys?
****  tokia, kokia surašyti.

*/

#include "hip/hip_runtime.h"

#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdio.h>

using namespace std;

#define INPUTFILE "LinkeviciusJ.txt"
#define CUDA_THREADS 10
#define MAX_DATA_PER_THREADS 10

/* Struktura saugoti vienai duomenu eilutei */
struct FileData
{
	char	stringField[255];
	int     intField;
	double  doubleField;

	FileData()
	{
		strncpy(stringField, "", 255);
		intField = 0;
		doubleField = 0.0;
	}

	FileData(string a, int b, double c)
	{
		strncpy(stringField, a.c_str(), 255);
		intField = b;
		doubleField = c;
	}
};

/* Struktura saugoti visiems duomenims */
struct ThreadData
{
	FileData array[ MAX_DATA_PER_THREADS ];
};

// lauku pavadinimai
string stringFieldName, intFieldName, doubleFieldName;

// nuskaito pradinius duomenis
void readData(ThreadData* threadDataArrays, int & threadDataSize, int & dataElementsCount)
{
	ifstream input(INPUTFILE);

	input >> stringFieldName;
	input >> intFieldName;
	input >> doubleFieldName;
	input >> dataElementsCount;

	threadDataSize = ceil((double)dataElementsCount / CUDA_THREADS);

	int line = 0;
	for (int i = 0; i < CUDA_THREADS; i++)
	{
		for (int j = 0; j < threadDataSize; j++)
		{
			string stringField;
			int intField;
			double doubleField;

			input >> stringField >> intField >> doubleField;

			// jei masyvui nebera duomenu, uzpildome tusciais elementais
			if (line < dataElementsCount)
				threadDataArrays[i].array[j] = FileData(stringField, intField, doubleField);
			else
				threadDataArrays[i].array[j] = FileData();

			line++;
		}
	}
	input.close();
}

// spausdina pradinius duomenis
void writeData(ThreadData* threadDataArrays, int & threadDataSize, int & dataElementsCount)
{
	int line = 0;
	cout << stringFieldName << "\t" << intFieldName << "\t" << doubleFieldName << "\r\n";
	for (int i = 0; i < CUDA_THREADS; i++)
	{
		cout << endl << "**** Array" << i << " ****" << endl;
		for (int j = 0; j < threadDataSize; j++)
		{
			line++;

			if (threadDataArrays[i].array[j].stringField != "")
			{
				cout.precision(2);
				cout << j << ") " << threadDataArrays[i].array[j].stringField << "\t" << threadDataArrays[i].array[j].intField << "\t" << fixed << threadDataArrays[i].array[j].doubleField << "\r\n";
			}

			if (line == dataElementsCount)
				break;
		}
	}
	cout << endl;
}

// lygiagrecioji programos dalis
// perduodamas duomenu masyvas ir kiekvieno proceso apdorojamu elementu kiekis
__global__ void printKernel(ThreadData *threadData, int* size)
{
	// get threadId
	int i = threadIdx.x;

	// spausdina proceso elementus
	for (int j = 0; j < *size; j++)
	{
		printf("process_%d: %d\t%s\t%d\t%.2f\n", i, j, threadData[i].array[j].stringField, threadData[i].array[j].intField, threadData[i].array[j].doubleField);
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t printWithCuda(ThreadData* hostData, int threadsCount, int threadDataSize)
{
	hipError_t cudaStatus;

	ThreadData* devData = 0;
	int*        devSize = 0;

	// choose gpu
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&devData, threadsCount * sizeof(ThreadData));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&devSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(devData, hostData, threadsCount * sizeof(ThreadData), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devSize, &threadDataSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	printf("Starting CUDA threads!\n");
	printKernel <<< 1, threadsCount >>> (devData, devSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	printf("End of CUDA threads!\n");

Error:
	hipFree(devData);
	hipFree(devSize);

	std::cin.get();

	return cudaStatus;
}

int main()
{
	ThreadData threadDataArrays[CUDA_THREADS];

	int dataElementsCount;
	int threadDataSize;

	readData(threadDataArrays, threadDataSize, dataElementsCount);

	writeData(threadDataArrays, threadDataSize, dataElementsCount);

	// Start CUDA
	hipError_t cudaStatus = printWithCuda(threadDataArrays, CUDA_THREADS, threadDataSize);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	std::cin.get();

	return 0;
}